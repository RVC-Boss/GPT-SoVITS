#include "hip/hip_runtime.h"
/* coding=utf-8
 * Copyright (c) 2024, NVIDIA CORPORATION.  All rights reserved.
 *
 * Licensed under the Apache License, Version 2.0 (the "License");
 * you may not use this file except in compliance with the License.
 * You may obtain a copy of the License at
 *
 *     http://www.apache.org/licenses/LICENSE-2.0
 *
 * Unless required by applicable law or agreed to in writing, software
 * distributed under the License is distributed on an "AS IS" BASIS,
 * WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 * See the License for the specific language governing permissions and
 * limitations under the License.
 */

#include <ATen/ATen.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_fp16.h>
#include <hip/hip_runtime_api.h>
#include <ATen/cuda/HIPContext.h>
#include <torch/extension.h>
#include "type_shim.h"
#include <assert.h>
#include <cfloat>
#include <limits>
#include <stdint.h>
#include <c10/macros/Macros.h>

namespace
{
    // Hard-coded hyperparameters
    // WARP_SIZE and WARP_BATCH must match the return values batches_per_warp and
    constexpr int ELEMENTS_PER_LDG_STG = 1; //(WARP_ITERATIONS < 4) ? 1 : 4;
    constexpr int BUFFER_SIZE = 32;
    constexpr int FILTER_SIZE = 12;
    constexpr int HALF_FILTER_SIZE = 6;
    constexpr int UPSAMPLE_REPLICATION_PAD = 5; // 5 on each side, matching torch impl
    constexpr int DOWNSAMPLE_REPLICATION_PAD_LEFT = 5; // matching torch impl
    constexpr int DOWNSAMPLE_REPLICATION_PAD_RIGHT = 6; // matching torch impl

    template <typename input_t, typename output_t, typename acc_t>
    __global__ void anti_alias_activation_forward(
        output_t *dst,
        const input_t *src,
        const input_t *up_ftr,
        const input_t *down_ftr,
        const input_t *alpha,
        const input_t *beta,
        int batch_size,
        int channels,
        int seq_len)
    {
        // Up and downsample filters
        input_t up_filter[FILTER_SIZE];
        input_t down_filter[FILTER_SIZE];

        // Load data from global memory including extra indices reserved for replication paddings
        input_t elements[2 * FILTER_SIZE + 2 * BUFFER_SIZE + 2 * UPSAMPLE_REPLICATION_PAD] = {0};
        input_t intermediates[2 * FILTER_SIZE + 2 * BUFFER_SIZE + DOWNSAMPLE_REPLICATION_PAD_LEFT + DOWNSAMPLE_REPLICATION_PAD_RIGHT] = {0};

        // Output stores downsampled output before writing to dst
        output_t output[BUFFER_SIZE];

        // blockDim/threadIdx = (128, 1, 1)
        // gridDim/blockIdx = (seq_blocks, channels, batches)
        int block_offset = (blockIdx.x * 128 * BUFFER_SIZE + seq_len * (blockIdx.y + gridDim.y * blockIdx.z));
        int local_offset = threadIdx.x * BUFFER_SIZE;
        int seq_offset = blockIdx.x * 128 * BUFFER_SIZE + local_offset;

        // intermediate have double the seq_len
        int intermediate_local_offset = threadIdx.x * BUFFER_SIZE * 2;
        int intermediate_seq_offset = blockIdx.x * 128 * BUFFER_SIZE * 2 + intermediate_local_offset;

        // Get values needed for replication padding before moving pointer
        const input_t *right_most_pntr = src + (seq_len * (blockIdx.y + gridDim.y * blockIdx.z));
        input_t seq_left_most_value = right_most_pntr[0];
        input_t seq_right_most_value = right_most_pntr[seq_len - 1];

        // Move src and dst pointers
        src += block_offset + local_offset;
        dst += block_offset + local_offset;

        // Alpha and beta values for snake activatons. Applies exp by default
        alpha = alpha + blockIdx.y;
        input_t alpha_val = expf(alpha[0]);
        beta = beta + blockIdx.y;
        input_t beta_val = expf(beta[0]);

        #pragma unroll
        for (int it = 0; it < FILTER_SIZE; it += 1)
        {
            up_filter[it] = up_ftr[it];
            down_filter[it] = down_ftr[it];
        }

        // Apply replication padding for upsampling, matching torch impl
        #pragma unroll
        for (int it = -HALF_FILTER_SIZE; it < BUFFER_SIZE + HALF_FILTER_SIZE; it += 1)
        {
            int element_index = seq_offset + it; // index for element
            if ((element_index < 0) && (element_index >= -UPSAMPLE_REPLICATION_PAD))
            {
                elements[2 * (HALF_FILTER_SIZE + it)] = 2 * seq_left_most_value;
            }
            if ((element_index >= seq_len) && (element_index < seq_len + UPSAMPLE_REPLICATION_PAD))
            {
                elements[2 * (HALF_FILTER_SIZE + it)] = 2 * seq_right_most_value;
            }
            if ((element_index >= 0) && (element_index < seq_len))
            {
                elements[2 * (HALF_FILTER_SIZE + it)] = 2 * src[it];
            }
        }

        // Apply upsampling strided convolution and write to intermediates. It reserves DOWNSAMPLE_REPLICATION_PAD_LEFT for replication padding of the downsampilng conv later
        #pragma unroll
        for (int it = 0; it < (2 * BUFFER_SIZE + 2 * FILTER_SIZE); it += 1)
        {
            input_t acc = 0.0;
            int element_index = intermediate_seq_offset + it; // index for intermediate
            #pragma unroll
            for (int f_idx = 0; f_idx < FILTER_SIZE; f_idx += 1)
            {
                if ((element_index + f_idx) >= 0)
                {
                    acc += up_filter[f_idx] * elements[it + f_idx];
                }
            }
            intermediates[it + DOWNSAMPLE_REPLICATION_PAD_LEFT] = acc;
        }

        // Apply activation function. It reserves DOWNSAMPLE_REPLICATION_PAD_LEFT and DOWNSAMPLE_REPLICATION_PAD_RIGHT for replication padding of the downsampilng conv later
        double no_div_by_zero = 0.000000001;
        #pragma unroll
        for (int it = 0; it < 2 * BUFFER_SIZE + 2 * FILTER_SIZE; it += 1)
        {
            intermediates[it + DOWNSAMPLE_REPLICATION_PAD_LEFT] += (1.0 / (beta_val + no_div_by_zero)) * sinf(intermediates[it + DOWNSAMPLE_REPLICATION_PAD_LEFT] * alpha_val) * sinf(intermediates[it + DOWNSAMPLE_REPLICATION_PAD_LEFT] * alpha_val);
        }

        // Apply replication padding before downsampling conv from intermediates
        #pragma unroll
        for (int it = 0; it < DOWNSAMPLE_REPLICATION_PAD_LEFT; it += 1)
        {
            intermediates[it] = intermediates[DOWNSAMPLE_REPLICATION_PAD_LEFT];
        }
        #pragma unroll
        for (int it = DOWNSAMPLE_REPLICATION_PAD_LEFT + 2 * BUFFER_SIZE + 2 * FILTER_SIZE; it < DOWNSAMPLE_REPLICATION_PAD_LEFT + 2 * BUFFER_SIZE + 2 * FILTER_SIZE + DOWNSAMPLE_REPLICATION_PAD_RIGHT; it += 1)
        {
            intermediates[it] = intermediates[DOWNSAMPLE_REPLICATION_PAD_LEFT + 2 * BUFFER_SIZE + 2 * FILTER_SIZE - 1];
        }

        // Apply downsample strided convolution (assuming stride=2) from intermediates
        #pragma unroll
        for (int it = 0; it < BUFFER_SIZE; it += 1)
        {
            input_t acc = 0.0;
            #pragma unroll
            for (int f_idx = 0; f_idx < FILTER_SIZE; f_idx += 1)
            {
                // Add constant DOWNSAMPLE_REPLICATION_PAD_RIGHT to match torch implementation
                acc += down_filter[f_idx] * intermediates[it * 2 + f_idx + DOWNSAMPLE_REPLICATION_PAD_RIGHT];
            }
            output[it] = acc;
        }

        // Write output to dst
        #pragma unroll
        for (int it = 0;  it < BUFFER_SIZE;  it += ELEMENTS_PER_LDG_STG)
        {
            int element_index = seq_offset + it;
            if (element_index < seq_len)
            {
                dst[it] = output[it];
            }
        }

    }

    template <typename input_t, typename output_t, typename acc_t>
    void dispatch_anti_alias_activation_forward(
        output_t *dst,
        const input_t *src,
        const input_t *up_ftr,
        const input_t *down_ftr,
        const input_t *alpha,
        const input_t *beta,
        int batch_size,
        int channels,
        int seq_len)
    {
        if (seq_len == 0)
        {
            return;
        }
        else
        {
            // Use 128 threads per block to maximimize gpu utilization
            constexpr int threads_per_block = 128;
            constexpr int seq_len_per_block = 4096;
            int blocks_per_seq_len = (seq_len + seq_len_per_block - 1) / seq_len_per_block;
            dim3 blocks(blocks_per_seq_len, channels, batch_size);
            dim3 threads(threads_per_block, 1, 1);

            anti_alias_activation_forward<input_t, output_t, acc_t>
                <<<blocks, threads, 0, at::cuda::getCurrentCUDAStream()>>>(dst, src, up_ftr, down_ftr, alpha, beta, batch_size, channels, seq_len);
        }
    }
}

extern "C" torch::Tensor fwd_cuda(torch::Tensor const &input, torch::Tensor const &up_filter, torch::Tensor const &down_filter, torch::Tensor const &alpha, torch::Tensor const &beta)
{
    // Input is a 3d tensor with dimensions [batches, channels, seq_len]
    const int batches = input.size(0);
    const int channels = input.size(1);
    const int seq_len = input.size(2);

    // Output
    auto act_options = input.options().requires_grad(false);

    torch::Tensor anti_alias_activation_results =
        torch::empty({batches, channels, seq_len}, act_options);

    void *input_ptr = static_cast<void *>(input.data_ptr());
    void *up_filter_ptr = static_cast<void *>(up_filter.data_ptr());
    void *down_filter_ptr = static_cast<void *>(down_filter.data_ptr());
    void *alpha_ptr = static_cast<void *>(alpha.data_ptr());
    void *beta_ptr = static_cast<void *>(beta.data_ptr());
    void *anti_alias_activation_results_ptr = static_cast<void *>(anti_alias_activation_results.data_ptr());

    DISPATCH_FLOAT_HALF_AND_BFLOAT(
        input.scalar_type(),
        "dispatch anti alias activation_forward",
        dispatch_anti_alias_activation_forward<scalar_t, scalar_t, float>(
            reinterpret_cast<scalar_t *>(anti_alias_activation_results_ptr),
            reinterpret_cast<const scalar_t *>(input_ptr),
            reinterpret_cast<const scalar_t *>(up_filter_ptr),
            reinterpret_cast<const scalar_t *>(down_filter_ptr),
            reinterpret_cast<const scalar_t *>(alpha_ptr),
            reinterpret_cast<const scalar_t *>(beta_ptr),
            batches,
            channels,
            seq_len););
    return anti_alias_activation_results;
}